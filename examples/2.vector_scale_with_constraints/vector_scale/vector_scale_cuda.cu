#include "hip/hip_runtime.h"
// CUDA component
#include <stdio.h>

static __global__ void vector_mult_cuda(float *val, unsigned n, float factor)
{
        unsigned i;
        for(i = 0 ; i < n ; i++)
               val[i] *= factor;
}



void scale_cuda_func(float *arr, int size, float factor)
{
  
    printf("\n************* Hello in vector-scale CUDA call ********************\n\n"); 
    
    vector_mult_cuda<<<1,1>>>(arr, size, factor);

	hipDeviceSynchronize();
}
